
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define CHECK_ERR(x)                                            \
    if (x != hipSuccess) {                                     \
        fprintf(stderr, "%s in %s at line %d\n",                \
                hipGetErrorString(err), __FILE__, __LINE__);   \
        exit(-1);                                               \
    }                                                           \

#define FILE_LINE 1024000
#define LINE_WIDTH 256

__device__ char *d_strstr(const char *str1, const char *str2, int width){
    char *cp = (char *)str1;
    char *s1, *s2;

    if(!*str2)
        return ((char*)str1);

    int i = 0;
    while(i < width)
    {
        s1 = cp;
        s2 = (char *)str2;

        while(*s1 && *s2 && !(*s1 - *s2))
            s1++, s2++;

        if(!*s2)
            return cp;
        cp++;
        i++;
    }

    return NULL;
}

__device__ char *d_memcpy(char *dest, char *src, int count)
{
    char *result = dest;

    if(dest <= src || dest >= (src + count))
    {
        while(count--)
            *(char *)dest++ = *(char *)src++;
    }
    else
    {
        dest += count - 1;
        src += count - 1;

        while(count--)
            *(char *)dest-- = *(char *)src--;
    }

    return result;
}

__global__ void d_Grep(char *d_File, char *d_regex, char *result, int line, int width){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    char *pch;
    if(i < line)
    {
        pch = d_strstr(&d_File[i*width], d_regex, width);
        if(pch != NULL)
            d_memcpy(&result[i*width], &d_File[i*width], sizeof(char)*width);
    }
}

int main(int argc, char* argv[])
{
    hipError_t err;

    char *Filename = argv[1];
	char *Regexp = argv[2];
    if(Regexp==NULL||Filename==NULL){
        printf("Usage: #./program [file name] [searching words]");
        return -1;
    }
    FILE *f;
    f = fopen(Filename, "r");
    if(f == NULL)
    {
        printf("Fail to open file!\n");
        return -1;
    }

    char *file;
    char *result;
    int i;

    file = (char *)malloc(sizeof(char)*FILE_LINE*LINE_WIDTH);
    result = (char *)malloc(sizeof(char)*FILE_LINE*LINE_WIDTH);

    fgets(file, FILE_LINE*LINE_WIDTH, f);

    char *d_file, *d_regex, *d_result;
    err = hipMalloc((void**) &d_file, sizeof(char)*FILE_LINE*LINE_WIDTH);
    CHECK_ERR(err);

    err = hipMalloc((void**) &d_regex, strlen(Regexp));
    CHECK_ERR(err);

    err = hipMalloc((void**) &d_result, sizeof(char)*FILE_LINE*LINE_WIDTH);
    CHECK_ERR(err);

    err = hipMemcpy(d_file, file, sizeof(char)*FILE_LINE*LINE_WIDTH, hipMemcpyHostToDevice);
    CHECK_ERR(err);

    err = hipMemcpy(d_regex, Regexp,  strlen(Regexp), hipMemcpyHostToDevice);
    CHECK_ERR(err);

    int numThread = 256;
    int numBlock = ceil((double)FILE_LINE/numThread);
    d_Grep<<<numBlock, numThread>>>(d_file, d_regex, d_result, FILE_LINE, LINE_WIDTH);

    err = hipMemcpy(result, d_result, sizeof(char)*FILE_LINE*LINE_WIDTH, hipMemcpyDeviceToHost);
    CHECK_ERR(err);

    for(i = 0; i < FILE_LINE; i++)
    {
        if(&result[i*LINE_WIDTH] != NULL)
            printf("%s", &result[i*LINE_WIDTH]);
    }

    return 0;
}
